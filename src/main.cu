#include "hip/hip_runtime.h"
__global__ void VecAdd(float *a, float *b, float *c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

int main() {
  VecAdd<<<1, N>>>(a, b, c);
  return 0;
}
